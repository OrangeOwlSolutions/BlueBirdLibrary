/*   Bluebird Library - High performance CPUs and GPUs computing library.
*    
*    Copyright (C) 2012-2013 Orange Owl Solutions.  
*
*    This file is part of Bluebird Library.
*    Bluebird Library is free software: you can redistribute it and/or modify
*    it under the terms of the Lesser GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    (at your option) any later version.
*
*    Bluebird Library is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    Lesser GNU General Public License for more details.
*
*    You should have received a copy of the GNU General Public License
*    along with Bluebird Library.  If not, see <http://www.gnu.org/licenses/>.
*
*
*    For any request, question or bug reporting please visit http://www.orangeowlsolutions.com/
*    or send an e-mail to: info@orangeowlsolutions.com
*
*
*/


#include "BB.h"
#include "TimingGPU.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct BB::PrivateTimingGPU {
		hipEvent_t		start;
		hipEvent_t		stop;
        };

// default constructor
BB::TimingGPU::TimingGPU() { privateTimingGPU = new BB::PrivateTimingGPU; }

// default destructor
BB::TimingGPU::~TimingGPU() { }

void BB::TimingGPU::StartCounter()
{
	hipEventCreate(&((*privateTimingGPU).start));
	hipEventCreate(&((*privateTimingGPU).stop));
	hipEventRecord((*privateTimingGPU).start,0);
}

void BB::TimingGPU::StartCounterFlags()
{
	int eventflags = hipEventBlockingSync;

	hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
	hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
	hipEventRecord((*privateTimingGPU).start,0);
}

// Gets the counter in ms
float BB::TimingGPU::GetCounter()
{
	float	time;
	hipEventRecord((*privateTimingGPU).stop, 0);
	hipEventSynchronize((*privateTimingGPU).stop);
	hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
	return time;
}

